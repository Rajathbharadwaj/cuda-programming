
#include <hip/hip_runtime.h>
#include <stdio.h>

//onCPU
void onCPU() 
{
	printf("This is running on CPU\n");
}

//Kernel runs on GPU
__global__ void onGPU()
{
	//keeps track of thread Index of the block
	int i = threadIdx.x;
	printf("This is running on GPU with the treadIndex of %d\n",&i);
}

int main() 
{
	//1 block/grid,  runs 5 threads/block
	onGPU<<<1, 5>>>();
	//Waits for GPU to finish 
	hipDeviceSynchronize();
	//runs on CPU normal c++ execution
	onCPU();
}
